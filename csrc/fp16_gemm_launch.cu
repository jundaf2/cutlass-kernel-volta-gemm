#include <iostream>
#include "fp16_gemm_kernel.cuh"
#include "fp16_gemm.h"

#include <cute/algorithm/copy.hpp>
#include <cute/algorithm/gemm.hpp>

#include <cutlass/cutlass.h>
#include <cutlass/array.h>
#include <cutlass/numeric_types.h>
#include <cutlass/numeric_conversion.h>
#include "cutlass/util/device_memory.h"
#include "cutlass/gemm/device/gemm.h"
#include "helper.h"

namespace volta {

// The code section below describes datatype for input, output matrices and computation between elements in input matrices.
using ElementAccumulator = float;                   // <- data type of accumulator
using ElementInputA = cutlass::half_t;              // <- data type of elements in input matrix A
using ElementInputB = cutlass::half_t;              // <- data type of elements in input matrix B
using ElementOutput = cutlass::half_t;              // <- data type of elements in output matrix D
using ElementComputeEpilogue = ElementAccumulator;  // <- data type of epilogue operations

// The code section below describes matrix layout of input and output matrices. Row Major for Matrix A, Row Major for Matrix B and Row Major for Matrix C
using LayoutInputA = cutlass::layout::RowMajor;
using LayoutInputB = cutlass::layout::RowMajor;
using LayoutOutput = cutlass::layout::RowMajor;

// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;
// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm70;
// This code section describes the tile size a thread block will compute
using ShapeMMAThreadBlock = cutlass::gemm::GemmShape<128, 128, 32>;  // <- threadblock tile M = 128, N = 128, K = 32
// This code section describes tile size a warp will compute
using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 32>;  // <- warp tile M = 64, N = 64, K = 32 
// This code section describes the size of MMA op
using ShapeMMAOp = cutlass::gemm::GemmShape<8, 8, 4>;  // <- MMA Op tile M = 8, N = 8, K = 4

using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                     // <- data type of output matrix
    128 / cutlass::sizeof_bits<ElementOutput>::value,  // <- this is the number of elements per vectorized memory access. For half precision, it's 8 elements. This becomes the vector width of math instructions in epilogue too
    ElementAccumulator,                                // <- data type of accumulator
    ElementComputeEpilogue>;  // <- data type for alpha/beta in linear combination function


// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>; 
// Number of pipelines you want to use
constexpr int NumStages = 2;


using Gemm = cutlass::gemm::device::Gemm<ElementInputA,
                                         LayoutInputA,
                                         ElementInputB,
                                         LayoutInputB,
                                         ElementOutput,
                                         LayoutOutput,
                                         ElementAccumulator,
                                         MMAOp,
                                         SmArch,
                                         ShapeMMAThreadBlock,
                                         ShapeMMAWarp,
                                         ShapeMMAOp,
                                         EpilogueOp,
                                         SwizzleThreadBlock,
                                         NumStages>;

void launch_matmul_kernel(Gemm_params &params, hipStream_t stream) {
    

    // Create a tuple of problem size for matrix multiplication
    cutlass::gemm::GemmCoord problem_size(params.M, params.N, params.K);

    typename cutlass::TensorRef<ElementInputA, LayoutInputA> tensor_a(reinterpret_cast<ElementInputA* >(params.a_ptr), params.K);
    typename cutlass::TensorRef<ElementInputB, LayoutInputB> tensor_b(reinterpret_cast<ElementInputB* >(params.b_ptr), params.N);
    typename cutlass::TensorRef<ElementOutput, LayoutOutput> tensor_c(reinterpret_cast<ElementOutput* >(params.c_ptr), params.N);

    // Initialize alpha and beta for dot product computation
    ElementComputeEpilogue alpha = ElementComputeEpilogue(1);
    ElementComputeEpilogue beta = ElementComputeEpilogue(0);

    // Split K dimension into 1 partitions
    int split_k_slices = 1;

    // Create a tuple of gemm kernel arguments. This is later passed as arguments to launch instantiated CUTLASS kernel
    typename Gemm::Arguments arguments{problem_size,  // <- problem size of matrix multiplication
                                        tensor_a,  // <- reference to matrix A on device
                                        tensor_b,  // <- reference to matrix B on device
                                        tensor_c,  // <- reference to matrix C on device
                                        tensor_c,  // <- reference to matrix D on device
                                        {alpha, beta},          // <- tuple of alpha and beta
                                        split_k_slices};        // <- k-dimension split factor

    // Instantiate CUTLASS kernel depending on templates
    Gemm gemm_op;
    // Check the problem size is supported or not 
    cutlass::Status status = gemm_op.can_implement(arguments);
    CUTLASS_CHECK(status);

    // Using the arguments, query for extra workspace required for matrix multiplication computation
    size_t workspace_size = Gemm::get_workspace_size(arguments);
    // Allocate workspace memory
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);
    std::cout << "workspace_size: " << workspace_size << std::endl;

    // Launch initialized CUTLASS kernel
    status = gemm_op(arguments);
    CUTLASS_CHECK(status);

    // constexpr int BM = 64;
    // constexpr int BN = 64;
    // constexpr int WM = 16;
    // constexpr int WN = 16;
    // constexpr int NUM_WARPS = (BM*BN)/(WM*WN);

    // const int BX = (params.N + BN - 1) / BN;
    // const int BY = (params.M + BM - 1) / BM;
    // dim3 grid(BX, BN);
    // cutlass::gemm::warp::WarpSize<typename OperatorClass>
    // dim3 block();
    // kernel::matmul_kernel<<<,,0,stream>>>(params, stream);
}
    
void run_gemm_fp16(Gemm_params &params, hipStream_t stream) {
    if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 1))) {
        std::cerr << "Volta Tensor Core operations must be compiled with CUDA 10.1 Toolkit or later." << std::endl;

        // Returning zero when built on older Toolkits so tests pass. The actions of this SDK example are no-op.
        return;
    }

    hipDeviceProp_t props;
    hipError_t error = hipGetDeviceProperties(&props, 0);
    if (error != hipSuccess) {
        std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
        return;
    }

    if (props.major != 7) {
        std::cerr << "Volta Tensor Ops must be run on a machine with compute capability of 70, 72, or 75." << std::endl;     
        return;   
    }

    launch_matmul_kernel(params, stream);
}
}